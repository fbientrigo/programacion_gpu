
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10

__global__ void modificarVariableGlobal(int* devVar)
{
    // modificar los valores
    devVar[threadIdx.x] += 2;
}

int main(void)
{
    // inicializar variable global
    int* hostVar = (int *) malloc(N*sizeof(int));
    int* devVar;
    hipMalloc((int**)&devVar, N*sizeof(int));

    for (int i = 0; i < N; i++){
        hostVar[i] = i;
    }

    // copiar valores al device (hay que usar "cudaMemcpyToSymbol")
    hipMemcpy(devVar, hostVar, N*sizeof(int), hipMemcpyHostToDevice);
    for (int i = 0; i < N; i++){
        printf("Antes del kernel: %d\n", hostVar[i]);
    }

    // invocar el kernel
    modificarVariableGlobal<<<1, N>>>(devVar);

    // copiar valores del device al host
    hipMemcpy(hostVar, devVar, N*sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++){
        printf("Después del kernel: %d\n", hostVar[i]);
    }

    hipDeviceReset();
    return EXIT_SUCCESS;
}
