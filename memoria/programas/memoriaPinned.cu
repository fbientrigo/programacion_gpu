
#include <hip/hip_runtime.h>
int main() {

    // cantidad de datos
    unsigned int N = 1<<22;
    unsigned int nbytes = N * sizeof(float);

    // asignar memoria pinned en el host
    float *h_a;
    hipHostMalloc((float **)&h_a, nbytes, hipHostMallocDefault);

    // asignar memoria en el device
    float *d_a;
    hipMalloc((float **)&d_a, nbytes);

    // inicializar datos en el host
    for(unsigned int i=0;i<N;i++) h_a[i] = 0.5f;

    // transferir datos del host al device
    hipMemcpy(d_a, h_a, nbytes, hipMemcpyHostToDevice);

    // transferir datos del device al host
    hipMemcpy(h_a, d_a, nbytes, hipMemcpyDeviceToHost);

    // liberar memoria
    hipFree(d_a);
    hipHostFree(h_a);

    // reset device
    hipDeviceReset();
    return EXIT_SUCCESS;
}
