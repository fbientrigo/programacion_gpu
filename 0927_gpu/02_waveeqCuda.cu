#include "hip/hip_runtime.h"
/*
CUDA Wave Equation Solver
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// Kernel CUDA para actualizar la ecuación de onda
__global__ void updateWave(double* un, double* un_m1, double* un_p1, int N, double delta_x, double delta_t, double c) {
    // Calcula el ID global del hilo
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i > 0 && i < N - 1) {
        // Coeficiente CFL para la ecuación de onda
        double cfl = c * c * delta_t * delta_t / (delta_x * delta_x);
        // Actualización usando el método de punto medio
        un_p1[i] = cfl * (un[i + 1] + un[i - 1] - 2.0 * un[i]) - un_m1[i] + 2.0 * un[i];
    }
}

// Inicializa los arreglos un, un_m1 y un_p1
void initialiseArrays(double* un, double* un_m1, double* un_p1, int N, double delta_x, double delta_t, double t, double c) {
    for (int i = 0; i < N; i++) {
        double xpos = i * delta_x;

        // Inicialización necesaria con una solución conocida (gaussiana)
        un[i] = gaussian(xpos, t, c);
        un_p1[i] = 0.0;
        un_m1[i] = gaussian(xpos, t - delta_t, c);
    }
}

int main() {
    double c, delta_t, delta_x, t_ini;

    c = 1.0;
    delta_t = 0.1;
    delta_x = 1.0;

    t_ini = 0.0;

    // Coeficiente CFL para la ecuación de onda
    double cfl = c * c * delta_t * delta_t / (delta_x * delta_x);

    int Ni = 10000;
    int Nt = 20000;
    int array_size = sizeof(double) * Ni;

    int count = 0;

    double* un;
    double* un_p1;
    double* un_m1;
    double* tmp;

    // Asignación de memoria en el dispositivo (GPU) utilizando Unified Memory
    hipMallocManaged(&un, array_size);
    hipMallocManaged(&un_p1, array_size);
    hipMallocManaged(&un_m1, array_size);

    char filename[20];

    // Inicializa los arreglos
    initialiseArrays(un, un_m1, un_p1, Ni, delta_x, delta_t, t_ini, c);

    for (int t = 0; t < Nt; t++) {
        if (t % 1000 == 0) {
            sprintf(filename, "u_%05d.dat", count);
            count += 1;

            FILE* datafile = fopen(filename, "w");
            fwrite(&(un[0]), sizeof(double), Ni, datafile);
            fclose(datafile);
        }

        // Llama al kernel CUDA para actualizar la ecuación de onda
        updateWave<<<(Ni + 255) / 256, 256>>>(un, un_m1, un_p1, Ni, delta_x, delta_t, c);

        // Intercambia los arreglos para la próxima iteración
        tmp = un;
        un_m1 = un;
        un = un_p1;
        un_p1 = tmp;
    }

    // Libera la memoria asignada en el dispositivo (GPU)
    hipFree(un);
    hipFree(un_p1);
    hipFree(un_m1);

    return 0;
}
